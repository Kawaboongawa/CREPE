#include "hip/hip_runtime.h"
#include "filters.cuh"


__global__ void set_cyan_kernel(
	uchar3* frame,
	const uint frame_resolution)
{
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	while (index < frame_resolution)
	{
		frame[index].x = 0;
		frame[index].y = 100;
		frame[index].z = 130;
		index += blockDim.x * gridDim.x;
	}
}

__global__ void swap_rb_kernel(const cv::cuda::PtrStepSz<uchar3> src, cv::cuda::PtrStep<uchar3> dst)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x < src.cols && y < src.rows)
	{
		uchar3 v = src(y, x);
		dst(y, x) = make_uchar3(v.z, v.y, v.x);
	}
}

__global__ void remove_b_kernel(cv::cuda::PtrStepSz<uchar3> src)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x < src.cols && y < src.rows)
	{
		uchar3 v = src(y, x);
		//NB: In opencv channel order is BGR
		src(y, x) = make_uchar3(0, v.y, v.z);
	}
}

void swap_rb_caller(const cv::cuda::PtrStepSz<uchar3>& src, cv::cuda::PtrStep<uchar3> dst)
{
	unsigned int threads_2d = get_max_threads_2d();
	dim3 threads(threads_2d, threads_2d);
	dim3 blocks((src.cols + threads_2d - 1) / threads_2d, (src.rows + threads_2d - 1) / threads_2d); 
	swap_rb_kernel <<<blocks, threads, 0, 0>>> (src, dst);
}

void remove_b_caller(cv::cuda::PtrStepSz<uchar3> src)
{
	unsigned int threads_2d = get_max_threads_2d();
	dim3 threads(threads_2d, threads_2d);
	dim3 blocks((src.cols + threads_2d - 1) / threads_2d, (src.rows + threads_2d - 1) / threads_2d);
	remove_b_kernel <<<blocks, threads, 0, 0>>> (src);
}


void set_cyan_caller(
	void *	    frame,
	const uint  frame_resolution)
{
	uint threads = get_max_threads();
	uint blocks = map_blocks_to_problem(frame_resolution, threads);
	set_cyan_kernel << <blocks, threads, 0, 0 >> > (static_cast<uchar3 *>(frame), frame_resolution);
}